#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <assert.h>
#include <algorithm>

#include "gdsync/device.cuh"

#include "gpu.h"

//---------------------------
// kernel stuff

__global__ void void_kernel()
{
        __threadfence_system();
        __syncthreads();
}

int gpu_launch_void_kernel_on_stream(hipStream_t s)
{
        const int nblocks = 1;
        const int nthreads = 1;
	void_kernel<<<nblocks, nthreads, 0, s>>>();
        CUDACHECK(hipGetLastError());
        return 0;
}

//----------

__global__ void dummy_kernel(int p0, float p1, float *p2)
{
        //const uint tid = threadIdx.x;
        //const uint bid = blockIdx.x;
        //const uint block_size = blockDim.x;
        //const uint grid_size = gridDim.x;
        //const uint gid = tid + bid*block_size;
        //const uint n_threads = block_size*grid_size;
        __syncthreads();
}

int gpu_launch_dummy_kernel(void)
{
        const int nblocks = over_sub_factor * gpu_num_sm;
        const int nthreads = 32;
        int p0 = 100;
        float p1 = 1.1f;
        float *p2 = NULL;
	dummy_kernel<<<nblocks, nthreads, 0, gpu_stream>>>(p0, p1, p2);
        CUDACHECK(hipGetLastError());
        return 0;
}

//----------

__global__ void calc_kernel(int n, float c, float *in, float *out)
{
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        const int block_size = blockDim.x;
        const int grid_size = gridDim.x;
        const int gid = tid + bid*block_size;
        const int n_threads = block_size*grid_size;
        for (int i=gid; i<n; i += n_threads)
                out[i] = in[i] * c;
}

int gpu_launch_calc_kernel_on_stream(size_t size, hipStream_t s)
{
        const int nblocks = over_sub_factor * gpu_num_sm;
        const int nthreads = 32*2;
        // at least one 1 float
        int n = (size+sizeof(float)-1) / sizeof(float);
        static float *in = NULL;
        static float *out = NULL;
        if (!in) {
                in = (float*)gpu_malloc(4096, n*sizeof(float));
                out = (float*)gpu_malloc(4096, n*sizeof(float));
        }
        // at least 1 thr block
        int nb = std::min(((n + nthreads - 1) / nthreads), nblocks);
        assert(nb >= 1);
	calc_kernel<<<nb, nthreads, 0, s>>>(n, 1.0f, in, out);
        CUDACHECK(hipGetLastError());
        return 0;
}

//----------

/*
 * Local variables:
 *  c-indent-level: 8
 *  c-basic-offset: 8
 *  tab-width: 8
 *  indent-tabs-mode: nil
 * End:
 */
