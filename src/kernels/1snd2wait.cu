#include "hip/hip_runtime.h"
/* Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "gdsync/device.cuh"
#include "objs.hpp"
#include "utils.hpp"
#include "gdsync_kernels.hpp"

using namespace gdsync;

extern "C" __global__ void krn1snd2wait(param_1snd2wait p)
{
    if (threadIdx.x==32) {
        device::release(p.sem0);
        //__threadfence_system();
        __threadfence();
        device::release(p.sem1);
    }

    // not stricly needed because there are two separate flows, doorbell
    // ringing for sends and notification of completions:
    //__syncthreads();

    else if (threadIdx.x<2) {
        device::wait(p.semw[threadIdx.x], p.condw[threadIdx.x]);
        device::release(p.sem23[threadIdx.x]);
    }
}

/*
 * Local variables:
 *  c-indent-level: 8
 *  c-basic-offset: 8
 *  tab-width: 8
 *  indent-tabs-mode: nil
 * End:
 */
